// thrust_sort_wrapper.cu
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/tuple.h>
// Define the EventQueueItem structure
extern "C" struct EventQueueItem {
    unsigned id;
    int material;
    float E;

    // Operator for comparison, used by thrust::sort
    __host__ __device__ bool operator<(const EventQueueItem& rhs) const {
        return thrust::tie(material, E) < thrust::tie(rhs.material, rhs.E);
    }
};

extern "C" void thrust_sort_event_queue(EventQueueItem* data, int n) {
    thrust::device_ptr<EventQueueItem> d_ptr(data);
    thrust::sort(thrust::device, d_ptr, d_ptr + n);
    hipDeviceSynchronize();
}

extern "C" void thrust_sort_float_array(float* data, int n) {
    // Create a device pointer for the input array
    thrust::device_ptr<float> d_ptr(data);
    // Sort the array in ascending order
    thrust::sort(thrust::device, d_ptr, d_ptr + n);
    hipDeviceSynchronize();
}

extern "C" void thrust_sort_int_array(int* data, int n) {
    thrust::device_ptr<int> d_ptr(data);
    thrust::sort(thrust::device, d_ptr, d_ptr + n);
    hipDeviceSynchronize();
}
